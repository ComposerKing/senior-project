#include "hip/hip_runtime.h"
#include "cnn_prim_device.hpp"

__global__
void vector_add_device(const float* __restrict__ a,
     		       const float* __restrict__ b,
		       float* __restrict__ c,
		       int size){
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if(i < size) 
   	c[i] = a[i] + b[i];
}