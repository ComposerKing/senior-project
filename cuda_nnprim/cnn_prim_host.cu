#define CNN_PRIM_TIMING

#include "cnn_prim_device.hpp"
#include <iostream>
#include <string>

#ifdef CNN_PRIM_TIMING
  #include <chrono>
#endif

// argument must be of type hipError_t!
#define CHECK_CUDA_ERROR(X) do{								    \
	hipError_t err = X;								    \
     	if(err){									    \
	  std::cerr << "CUDA Error at line " << __LINE__ << " in file " << __FILE__ << ": " \
	  << hipGetErrorName(err) << " -- " << hipGetErrorString(err) << std::endl;  	    \
	}										    \
     }while(0)


__host__
void vector_add_cuda(const float* h_a, const float* h_b, float* h_c, const int& size){

     //GPU work specifications
     int blockDim = 256;
     int gridDim = (size + blockDim-1) / blockDim;
     
     float *d_a, *d_b, *d_c;

     //Allocate GPU memory
     CHECK_CUDA_ERROR(hipMalloc((void**)&d_a, size*sizeof(float)));
     CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, size*sizeof(float)));
     CHECK_CUDA_ERROR(hipMalloc((void**)&d_c, size*sizeof(float)));

     //Transfer to GPU
     CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, size*sizeof(float), hipMemcpyHostToDevice));
     CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, size*sizeof(float), hipMemcpyHostToDevice));


#ifdef CNN_PRIM_TIMING
     std::chrono::high_resolution_clock::time_point start;
     std::chrono::high_resolution_clock::time_point end;
     std::chrono::duration<double> elapsed;
    
     start = std::chrono::high_resolution_clock::now();
#endif

     //Kernel call & block
     vector_add_device<<<gridDim, blockDim>>>(d_a, d_b, d_c, size);
     CHECK_CUDA_ERROR(hipDeviceSynchronize());

#ifdef CNN_PRIM_TIMING
     end = std::chrono::high_resolution_clock::now();
     elapsed = std::chrono::duration_cast<std::chrono::duration<double>>(end-start);
     std::cout << "\"vector_add_cuda\" kernel took " << elapsed.count() << " secs." << std::endl;
#endif

     //Get answer
     CHECK_CUDA_ERROR(hipMemcpy(h_c, d_c, size*sizeof(float), hipMemcpyDeviceToHost));

     //Release device allocations and reset
     CHECK_CUDA_ERROR(hipFree(d_a));
     CHECK_CUDA_ERROR(hipFree(d_b));
     CHECK_CUDA_ERROR(hipFree(d_c));
     CHECK_CUDA_ERROR(hipDeviceReset());
}
